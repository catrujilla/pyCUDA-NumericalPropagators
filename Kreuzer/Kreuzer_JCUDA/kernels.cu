
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
typedef unsigned char Pixel;

extern "C"
__global__ void
fft_shift(float *real, float *imag, float *d_temp13x, int width, int height)
{

	/*Variables que nos delimitan la operacion en CUDA (Para no coger todos los hilos disponibles,
	ya que esto implica m�s tiempo de operaci�n en CUDA*/

	int m2 = width / 2;
	int n2 = height / 2;

	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;
	int col2 = blockIdx.x*blockDim.x + threadIdx.x + m2;
	int fila2 = blockIdx.y*blockDim.y + threadIdx.y + n2;

	/*
	Dibujo esquematico de cada cuadrante:

	////////////////////////////////////////////////////////////////////////////////////
	/                                         /                                        /
	/                    /                    /             ///                        /
	/                   //                    /            /   /                       /
	/                  / /                    /                /                       /
	/                 /  /                    /               /                        /
	/                    /                    /              /                         /
	/                    /                    /             /                          /
	/                    /                    /            /////                       /
	/                                         /                                        /
	////////////////////////////////////////////////////////////////////////////////////
	/                                         /                                        /
	/                   /                     /               ////                     /
	/                  //                     /                  /                     /
	/                /  /                     /                  /                     /
	/              //////                     /               ////                     /
	/                   /                     /                  /                     /
	/                   /                     /                  /                     /
	/                   /                     /               ////                     /
	/                                         /                                        /
	////////////////////////////////////////////////////////////////////////////////////

	*/

	/*Este condicional limita los hilos que se encargan de llevar a cabo el proceso, no es necesario por la simetr�a
	(matrix de N*N), pero en caso de no existir esta simetr�a es fundamental*/

	//if (col2 < m2 && col < m2 && fila2 < n2 && fila < m2) {   

	d_temp13x[fila*width + col] = real[fila*width + col];  //Guardo el primer cuadrante
	real[fila*width + col] = real[fila2*width + col2];  //en el primer cuadrante estoy poniendo lo que hay en el tercero
	real[fila2*width + col2] = d_temp13x[fila*width + col];//En el tercer cuadrante estoy poniendo lo que habia en el primero

	d_temp13x[fila*width + col] = imag[fila*width + col];  //Lo mismo anterior pero para los imaginarios
	imag[fila*width + col] = imag[fila2*width + col2];
	imag[fila2*width + col2] = d_temp13x[fila*width + col];

	d_temp13x[fila*width + col] = real[fila*width + col2];//Guardo Cuadrante dos
	real[fila*width + col2] = real[fila2*width + col];  //En el segundo guardo lo que hay en el cuarto
	real[fila2*width + col] = d_temp13x[fila*width + col];//En el cuarto guardo lo que estaba en el segundo

	d_temp13x[fila*width + col] = imag[fila*width + col2]; //Lo mismo que en el anterior
	imag[fila*width + col2] = imag[fila2*width + col];
	imag[fila2*width + col] = d_temp13x[fila*width + col];
	//}

}

extern "C"
__global__ void
Umbralizacion(Pixel *odata, float *temp, float umbral, int width, int height)
{
	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	//Calculo de la intensidad
	if (temp[fila*width + col] >= umbral) {
		temp[fila*width + col] = temp[fila*width + col];
	}
	else {
		temp[fila*width + col] = 0;
	}
	__syncthreads();
	odata[fila*width + col] = (char)((temp[fila*width + col]));


}

extern "C"
__global__ void
Ventana(float *odata, float *temporal1, int width, int height, int ancho, int posicionXInt, int posicionYInt)
{
	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	//Calculo de la intensidad
	if (col > (posicionXInt - ancho) && col < (posicionXInt + ancho) &&
		fila >(posicionYInt - ancho) && fila < (posicionYInt + ancho)) {
		temporal1[fila*width + col] = odata[fila*width + col];
		//temporal1[ fila*width+col ] = 255;
	}
	else {
		temporal1[fila*width + col] = 0;
	}

}

extern "C"
__global__ void
metrica(hipfftComplex *odata, float *temp_intensidad, int width, int height)
{
	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	//Calculo de la intensidad
	temp_intensidad[fila*width + col] = ((odata[fila*width + col].x)*(odata[fila*width + col].x));
	//temp_intensidad[ fila*width+col ] = sqrt(temp_intensidad[ fila*width+col ]);
	//temp_intensidad[ fila*width+col ] = temp_intensidad[ fila*width+col ] * temp_intensidad[ fila*width+col ];
	//temp_intensidad[ fila*width+col ]=__log10f(temp_intensidad[ fila*width+col ]);
	//Ac? tenemos todas las intensidades resultantes

}

extern "C"
__global__ void
modulo(float *odata_real, float *odata_imag, float *temp_intensidad, int width, int height)
{
	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	//Calculo de la intensidad
	temp_intensidad[fila*width + col] = ((odata_real[fila*width + col])*(odata_real[fila*width + col])) +
		((odata_imag[fila*width + col])*(odata_imag[fila*width + col]));
	temp_intensidad[fila*width + col] = sqrt(temp_intensidad[fila*width + col]);
	odata_real[fila*width + col] = temp_intensidad[fila*width + col] * temp_intensidad[fila*width + col];
	//temp_intensidad[ fila*width+col ]=__log10f(temp_intensidad[ fila*width+col ]);
	//Ac? tenemos todas las intensidades resultantes

}

extern "C"
__global__ void
amplitud(float *odata_real, float *odata_imag, float *temp_intensidad, int width, int height)
{
	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	//Calculo de la intensidad
	temp_intensidad[fila*width + col] = ((odata_real[fila*width + col])*(odata_real[fila*width + col])) +
		((odata_imag[fila*width + col])*(odata_imag[fila*width + col]));
		
	//odata_real[fila*width + col] = (temp_intensidad[fila*width + col]);
	odata_real[fila*width + col] = sqrt(temp_intensidad[fila*width + col]);

	//Ac? tenemos todas las intensidades resultantes

}

extern "C"
__global__ void
moduloLog10(float *odata_real, float *odata_imag, float *temp_intensidad, int width, int height)
{
	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	//Calculo de la intensidad
	temp_intensidad[fila*width + col] = ((odata_real[fila*width + col])*(odata_real[fila*width + col])) +
		((odata_imag[fila*width + col])*(odata_imag[fila*width + col]));
	temp_intensidad[fila*width + col] = sqrt(temp_intensidad[fila*width + col]);
	odata_real[fila*width + col] = temp_intensidad[fila*width + col] * temp_intensidad[fila*width + col];
	odata_real[fila*width + col] = __log10f(odata_real[fila*width + col]);
	
	//temp_intensidad[ fila*width+col ]=__log10f(temp_intensidad[ fila*width+col ]);
	//Ac? tenemos todas las intensidades resultantes

}

extern "C"
__global__ void
visualizar(Pixel *odata, float *temp_intensidad, int width, int height, float maximo, float minimo)
{

	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	//Ac? tenemos todas los pixeles escalados a 8 bits (255 niveles de gris)
	//temporal1[ fila*width+col ] = (temp_intensidad[ fila*width+col ])*255;  //C?lculo de Coeficiente de Tamura
	odata[fila*width + col] = (char)((temp_intensidad[fila*width + col]) * 1);
}


extern "C"
__global__ void
escalamiento(float *temp, int width, int height, float maximo, float minimo)
{

	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	temp[fila*width + col] = (temp[fila*width + col]) - minimo;
	temp[fila*width + col] = (temp[fila*width + col]) / (maximo - minimo);
	temp[fila*width + col] = (temp[fila*width + col]) * 255;
	//Ac? tenemos todas los pixeles escalados a 8 bits (255 niveles de gris)

}

extern "C"
__global__ void
Restando_Referencia(float *real, float *temp, int width, int height){

	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	//Haciendo la resta...
	real[fila*width + col] = real[fila*width + col] - temp[fila*width + col];
	temp[fila*width + col] = 0;

}

extern "C"
__global__ void
Restando_Promedio(hipfftComplex *idata, float *temporal1, float promedio, int width, int height){

	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;


	//Haciendo la resta...
	idata[fila*width + col].x = temporal1[fila*width + col];// - promedio;// - idata_R[ fila*width+col ];

}

extern "C"
__global__ void
Kreuzer_Remapeo(float *idata, float *idata_remap, int width, int height, float parametro_d, float L, float W)
{

	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	/*Ac? aplicamos la trasformaci?n planteada por kreuzer para el holograma*/

	float Xo, Yo, deltaX, deltaY;
	float XoP, YoP, deltaXP, deltaYP;
	float L2;

	L2 = L*L;

	Xo = -W / 2.0;
	Yo = W / 2.0;

	deltaX = W / (width - 1);
	deltaY = -W / (height - 1);

	XoP = Xo * L / sqrt(L2 + Xo*Xo);
	YoP = Yo * L / sqrt(L2 + Xo*Xo);

	deltaXP = -2.0 * XoP / (width - 1);
	deltaYP = -2.0 * YoP / (height - 1);

	float deltaXm1 = 1.0 / deltaX;
	float deltaYm1 = 1.0 / deltaY;

	float YPos = YoP + fila*deltaYP;
	float XPos = XoP + col*deltaXP;
	float RPm1 = 1 / sqrt(L2 - XPos*XPos - YPos*YPos);

	float newXPos = XPos*L*RPm1;
	float newYPos = YPos*L*RPm1;

	float Xcoord = (newXPos - Xo)*deltaXm1;
	float Ycoord = (newYPos - Yo)*deltaYm1;

	int iXcoord = (int)floor(Xcoord);
	int iYcoord = (int)floor(Ycoord);

	float x1frac = (iXcoord + 1.0) - Xcoord;
	float x2frac = 1.0 - x1frac;
	float y1frac = (iYcoord + 1.0) - Ycoord;
	float y2frac = 1.0 - y1frac;

	float x1y1 = x1frac*y1frac;
	float x1y2 = x1frac*y2frac;
	float x2y1 = x2frac*y1frac;
	float x2y2 = x2frac*y2frac;

	/*
	//Inicializamos el array donde quedaran los datos del holograma remapeado
	idata_remap[ fila*width+col ].x = 0;
	*/

	/*
	idata[ fila*width+col ].x = idata[ fila*width+col ].x - promedio;

	*/

	//Teniendo todos los valores listos, ahora hacemos el "remapeo" sobre el holograma

	if (iYcoord>0 && iYcoord<height / 2 && iXcoord>0 && iXcoord<width / 2)
	{
		//Cuadrante 1
		idata_remap[fila*width + col] = ((x1y1*idata[iYcoord*width + iXcoord])
			+ (x2y1*idata[iYcoord*width + iXcoord + 1])
			+ (x1y2*idata[(iYcoord + 1)*width + iXcoord])
			+ (x2y2*idata[(iYcoord + 1)*width + iXcoord + 1]));

		//Cuadrante 2
		idata_remap[(fila + 1)*width - 1 - col] = ((x1y1*idata[(iYcoord + 1)*width - 1 - iXcoord])
			+ (x2y1*idata[(iYcoord + 1)*width - 1 - iXcoord - 1])
			+ (x1y2*idata[(iYcoord + 2)*width - 1 - iXcoord])
			+ (x2y2*idata[(iYcoord + 2)*width - 1 - iXcoord - 1]));

		//cudrante 3
		idata_remap[(height - 1 - fila)*width + col] = (x1y1*idata[(height - 1 - iYcoord)*width + iXcoord] +
			x2y1*idata[(height - 1 - iYcoord)*width + iXcoord + 1] +
			x1y2*idata[(height - 2 - iYcoord)*width + iXcoord] +
			x2y2*idata[(height - 2 - iYcoord)*width + iXcoord + 1]);

		//Cuadrante 4
		idata_remap[(height - fila)*width - col - 1] = (x1y1*idata[(height - iYcoord)*width - iXcoord - 1] +
			x2y1*idata[(height - iYcoord)*width - iXcoord - 2] +
			x1y2*idata[(height - 1 - iYcoord)*width - iXcoord - 1] +
			x2y2*idata[(height - 1 - iYcoord)*width - iXcoord - 2]);

	}

}

extern "C"
__global__ void
multiplicacion_fase(float *idata_real, float *idata_imag, float *odata_real, float *odata_imag, int width, int height, float parametro_d, float L, float W, float lambda)
{

	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	// Par?metros


	float deltax = W / width;

	float z = parametro_d;

	float L2 = L*L;

	float k = 2 * (3.141592) / lambda;

	float deltaX = (z)*deltax / L;
	float deltaY = deltaX;

	//origen de coordenadas holograma
	float xo = -W / 2;
	float yo = xo;

	//origen de coordenadas holograma transformado
	float xop = xo * L / sqrt(L2 + xo*xo);
	float yop = xop;

	float deltaxp = L / (width)* ((xo + (width - 1)*deltax) / sqrt(L2 + (xo + (width - 1)*deltax)*(xo + (width - 1)*deltax)) - xo / sqrt(L2 + xo*xo));
	float deltayp = deltaxp;

	//origen de coordenadas plano de reconstrucci?n
	float Yo = -deltaX*(width) / 2;
	float Xo = Yo;

	float termino_1 = (k / L)*((Xo + col*deltaX)*xop + (Yo + fila*deltaY)*yop);
	float termino_2 = (0.5*k / L)*((col - width / 2)*(col - width / 2)*deltaxp*deltaX + (fila - width / 2)*(fila - width / 2)*deltayp*deltaY);


	//Ac? empieza lo paralelo:
	float real = deltaxp*deltayp*((__cosf(termino_1) * __cosf(termino_2)) -
		(__sinf(termino_1) * __sinf(termino_2)));

	float imag = deltaxp*deltayp*((__cosf(termino_1) * __sinf(termino_2)) +
		(__sinf(termino_1) * __cosf(termino_2)));


	//Cuadrante 1
	odata_real[fila*width + col] = (idata_real[fila*width + col] * real) - (idata_imag[fila*width + col] * imag);
	odata_imag[fila*width + col] = (idata_imag[fila*width + col] * real) + (idata_real[fila*width + col] * imag);

	/*
	//Cuadrante 2
	idata_remap[(fila+1)*width-1-col].x = (idata[(fila+1)*width-1-col].x * real) - (idata[(fila+1)*width-1-col].y * imag);
	idata_remap[(fila+1)*width-1-col].y = (idata[(fila+1)*width-1-col].y * real) + (idata[(fila+1)*width-1-col].x * imag);


	//Cuadrante 3
	idata_remap[(height-1-fila)*width+col].x = (idata[(height-1-fila)*width+col].x * real) - (idata[(height-1-fila)*width+col].y * imag);
	idata_remap[(height-1-fila)*width+col].y = (idata[(height-1-fila)*width+col].y * real) + (idata[(height-1-fila)*width+col].x * imag);


	//Cuadrante 4
	idata_remap[(height-fila)*width-col-1].x = (idata[(height-fila)*width-col-1].x * real) - (idata[(height-fila)*width-col-1].y * imag);
	idata_remap[(height-fila)*width-col-1].y = (idata[(height-fila)*width-col-1].y * real) + (idata[(height-fila)*width-col-1].x * imag);
	*/


}

extern "C"
__global__ void
multiplicacion(float *idata_remap_real, float *idata_remap_imag, float *matriz_holo_real, float *matriz_holo_imag, float *d_temp13x, int width, int height)
{

	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	d_temp13x[fila*width + col] = ((idata_remap_real[fila*width + col])*(matriz_holo_real[fila*width + col])) -
		((idata_remap_imag[fila*width + col])*(matriz_holo_imag[fila*width + col]));

	matriz_holo_imag[fila*width + col] = ((idata_remap_imag[fila*width + col])*(matriz_holo_real[fila*width + col])) +
		((idata_remap_real[fila*width + col])*(matriz_holo_imag[fila*width + col]));

	matriz_holo_real[fila*width + col] = d_temp13x[fila*width + col];

}

extern "C"
__global__ void
CambioTipoVariable(float *idata_remap_real, float *idata_remap_imag, float *matriz_holo_real, float *matriz_holo_imag, float *arreglo1, float *arreglo2, int width, int height)
{

	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	arreglo1[(fila*width + col) * 2] = idata_remap_real[fila*width + col]; //En los pares meto los reales...
	arreglo1[((fila*width + col) * 2) + 1] = idata_remap_imag[fila*width + col]; //En los impares meto los imaginarios

	arreglo2[(fila*width + col) * 2] = matriz_holo_real[fila*width + col]; //En los pares meto los reales...
	arreglo2[((fila*width + col) * 2) + 1] = matriz_holo_imag[fila*width + col]; //En los impares meto los imaginarios

}

extern "C"
__global__ void
CambioTipoVariable2(float *idata_remap_real, float *idata_remap_imag, float *matriz_holo_real, float *matriz_holo_imag, float *arreglo1, float *arreglo2, int width, int height)
{

	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	idata_remap_real[fila*width + col] = arreglo1[(fila*width + col) * 2]; //En los pares meto los reales...
	idata_remap_imag[fila*width + col] = arreglo1[((fila*width + col) * 2) + 1]; //En los impares meto los imaginarios

	matriz_holo_real[fila*width + col] = arreglo2[(fila*width + col) * 2]; //En los pares meto los reales...
	matriz_holo_imag[fila*width + col] = arreglo2[((fila*width + col) * 2) + 1]; //En los impares meto los imaginarios

}

extern "C"
__global__ void
generacion_f1_f2(float *idata_remap_real, float *idata_remap_imag, float *matriz_holo_real, float *matriz_holo_imag, int width, int height, float parametro_d, float M, float L, float W, float lambda)
{

	//Descriptores de cada hilo
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int fila = blockIdx.y*blockDim.y + threadIdx.y;

	// Par?metros


	float deltax = W / width;

	float z = parametro_d;	

	float L2 = L*L;


	float k = 2 * (3.141592) / lambda;

	float deltaX = M*(z)*deltax / L;
	float deltaY = deltaX;

	//origen de coordenadas holograma
	float xo = -W / 2;
	float yo = xo;

	//origen de coordenadas holograma transformado
	float xop = xo * L / sqrt(L2 + xo*xo);
	float yop = xop;

	float deltaxp = L / (width)* ((xo + (width - 1)*deltax) / sqrt(L2 + (xo + (width - 1)*deltax)*(xo + (width - 1)*deltax)) - xo / sqrt(L2 + xo*xo));
	float deltayp = deltaxp;

	//origen de coordenadas plano de reconstrucci?n
	float Yo = -deltaX*(width) / 2;
	float Xo = -deltaX*(width) / 2;

	//MULTIPLICAR HOLOGRAMA POR FASE DE PROPAGACI?N
	float Rp = sqrt((L2)-((deltaxp*col + xop)*(deltaxp*col + xop)) - ((deltayp*fila + yop)*(deltayp*fila + yop)));

	float argumento_1 = (L / Rp)*(L / Rp)*(L / Rp)*(L / Rp);
	float argumento_2 = k*z*Rp / L;	


	float Ip_real = (argumento_1)*__cosf(argumento_2);
	float Ip_imag = (argumento_1)*__sinf(argumento_2);


	float fase_real = __cosf((k / (2 * L))*(2 * Xo*col*deltaxp + 2 * Yo*fila*deltayp + (col*col)*deltaxp*deltaX + (fila*fila)*deltayp*deltaY));
	float fase_imag = __sinf((k / (2 * L))*(2 * Xo*col*deltaxp + 2 * Yo*fila*deltayp + (col*col)*deltaxp*deltaX + (fila*fila)*deltayp*deltaY));


	//funci?n f1
	//Cuadrante 1
	idata_remap_imag[fila*width + col] = ((idata_remap_real[fila*width + col] * Ip_real*fase_imag) + (idata_remap_real[fila*width + col]*Ip_imag*fase_real));
	idata_remap_real[fila*width + col] = ((idata_remap_real[fila*width + col] * Ip_real*fase_real) - (idata_remap_real[fila*width + col]*Ip_imag*fase_imag));

	/*
	//Cuadrante 2
	idata_remap[(fila+1)*width-1-col].y = ((idata_remap[(fila+1)*width-1-col].x*Ip_real*fase_imag)+(idata_remap[(fila+1)*width-1-col].x*Ip_imag*fase_real));
	idata_remap[(fila+1)*width-1-col].x = ((idata_remap[(fila+1)*width-1-col].x*Ip_real*fase_real)-(idata_remap[(fila+1)*width-1-col].x*Ip_imag*fase_imag));


	//Cuadrante 3
	idata_remap[(height-1-fila)*width+col].y = ((idata_remap[(height-1-fila)*width+col].x*Ip_real*fase_imag)+(idata_remap[(height-1-fila)*width+col].x*Ip_imag*fase_real));
	idata_remap[(height-1-fila)*width+col].x = ((idata_remap[(height-1-fila)*width+col].x*Ip_real*fase_real)-(idata_remap[(height-1-fila)*width+col].x*Ip_imag*fase_imag));


	//Cuadrante 4
	idata_remap[(height-fila)*width-col-1].y = ((idata_remap[(height-fila)*width-col-1].x*Ip_real*fase_imag)+(idata_remap[(height-fila)*width-col-1].x*Ip_imag*fase_real));
	idata_remap[(height-fila)*width-col-1].x = ((idata_remap[(height-fila)*width-col-1].x*Ip_real*fase_real)-(idata_remap[(height-fila)*width-col-1].x*Ip_imag*fase_imag));
	*/

	//funcion f2
	//Cuadrante 1
	matriz_holo_real[fila*width + col] = __cosf((k / (2 * L))*((col - width / 2)*(col - width / 2)*deltaxp*deltaX + (fila - width / 2)*(fila - width / 2)*deltayp*deltaY));
	matriz_holo_imag[fila*width + col] = (-1)*__sinf((k / (2 * L))*((col - width / 2)*(col - width / 2)*deltaxp*deltaX + (fila - width / 2)*(fila - width / 2)*deltayp*deltaY));

	/*
	//Cuadrante 2
	matriz_holo[(fila+1)*width-1-col].x = matriz_holo[fila*width+col].x;
	matriz_holo[(fila+1)*width-1-col].y = matriz_holo[fila*width+col].y;


	//Cuadrante 3
	matriz_holo[(height-1-fila)*width+col].x = matriz_holo[fila*width+col].x;
	matriz_holo[(height-1-fila)*width+col].y = matriz_holo[fila*width+col].y;


	//Cuadrante 4
	matriz_holo[(height-fila)*width-col-1].x = matriz_holo[fila*width+col].x;
	matriz_holo[(height-fila)*width-col-1].y = matriz_holo[fila*width+col].y;
	*/
}

extern "C"
__global__ void Obtener_Promedio(float *pArray, float *pAvgResults)
{
	// Declare arrays to be in shared memory.
	// 128 elements * (4 bytes / element) * 2 = 2KB.
	__shared__ float avg[256];

	// Calculate which element this thread reads from memory
	int arrayIndex = 256 * 128 * blockIdx.y + 256 * blockIdx.x + threadIdx.x;
	avg[threadIdx.x] = pArray[arrayIndex];
	__syncthreads();


	int nTotalThreads = blockDim.x;	// Total number of active threads

	while (nTotalThreads > 1)
	{
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
		if (threadIdx.x < halfPoint)
		{
			// when calculating the average, sum and divide
			avg[threadIdx.x] += avg[threadIdx.x + halfPoint];
			avg[threadIdx.x] /= 2;
		}

		nTotalThreads = (nTotalThreads >> 1);	// divide by two.
	}

	if (threadIdx.x == 0)
	{
		pAvgResults[128 * blockIdx.y + blockIdx.x] = avg[0];

	}

}

extern "C"
__global__ void Sumatoria(float *pArray, float *pDesviacion)
{
	// Declare arrays to be in shared memory.
	// 128 elements * (4 bytes / element) * 2 = 2KB.
	__shared__ float avg[256];

	// Calculate which element this thread reads from memory
	int arrayIndex = 256 * 128 * blockIdx.y + 256 * blockIdx.x + threadIdx.x;
	avg[threadIdx.x] = pArray[arrayIndex];
	__syncthreads();


	int nTotalThreads = blockDim.x;	// Total number of active threads

	while (nTotalThreads > 1)
	{
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
		if (threadIdx.x < halfPoint)
		{
			// when calculating the average, sum and divide
			avg[threadIdx.x] += avg[threadIdx.x + halfPoint];
			//avg[threadIdx.x] /= 2;
		}

		nTotalThreads = (nTotalThreads >> 1);	// divide by two.
	}

	if (threadIdx.x == 0)
	{
		pDesviacion[128 * blockIdx.y + blockIdx.x] = avg[0];

	}

}

extern "C"
__global__ void getStats(float *pArray, float *pMaxResults, float *pMinResults)
{
	// Declare arrays to be in shared memory.
	// 256 elements * (4 bytes / element) * 2 = 2KB.
	__shared__ float min[256];
	__shared__ float max[256];

	// Calculate which element this thread reads from memory
	int arrayIndex = 256 * 128 * blockIdx.y + 256 * blockIdx.x + threadIdx.x;
	min[threadIdx.x] = max[threadIdx.x] = pArray[arrayIndex];
	__syncthreads();


	int nTotalThreads = blockDim.x;	// Total number of active threads

	while (nTotalThreads > 1)
	{
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
		if (threadIdx.x < halfPoint)
		{
			// Get the shared value stored by another thread
			float temp = min[threadIdx.x + halfPoint];
			if (temp < min[threadIdx.x]) min[threadIdx.x] = temp;
			temp = max[threadIdx.x + halfPoint];
			if (temp > max[threadIdx.x]) max[threadIdx.x] = temp;
		}


		nTotalThreads = (nTotalThreads >> 1);	// divide by two.
	}

	// At this point in time, thread zero has the min, max, and average
	// It's time for thread zero to write it's final results.
	// Note that the address structure of pResults is different, because
	// there is only one value for every thread block.

	if (threadIdx.x == 0)
	{
		pMaxResults[128 * blockIdx.y + blockIdx.x] = max[0];
		pMinResults[128 * blockIdx.y + blockIdx.x] = min[0];

	}
}
